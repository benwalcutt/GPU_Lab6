
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

// Define your kernels in this file you may use more than one kernel if you
// need to

#define BLOCK_SIZE 512
#define NUM_BLOCKS 30
#define OFFSET (BLOCK_SIZE * NUM_BLOCKS)

// INSERT KERNEL(S) HERE

__global__ void histogram_kernel(unsigned int* input, unsigned int* bins,
    unsigned int num_elements, unsigned int num_bins) {
		
	extern __shared__ unsigned int bins_s[];
	
	for (int j = 0; j < num_bins; j++) {
		bins_s[j] = 0;
	}
		
	int globalId = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	unsigned int temp;
	
	// determine how many times are assigned grid is supposed to run
	int iterations = num_elements / (BLOCK_SIZE * NUM_BLOCKS) + 1;
	
	for (int i = 0; i < iterations; i++) {
		if (globalId + (OFFSET * i) < num_elements) {
			temp = input[globalId + (OFFSET * i)];
			temp = atomicAdd(&bins_s[temp], 1u);
		}
	}
	__syncthreads();
	
	iterations = num_bins / (BLOCK_SIZE) + 1;
	
	for (int i = 0; i < iterations; i++) {
		if (threadIdx.x + (i * BLOCK_SIZE) < num_bins) {
			temp = bins_s[threadIdx.x + (i*BLOCK_SIZE)];
			temp = atomicAdd(&bins[threadIdx.x + (i*BLOCK_SIZE)], temp);
		}
	}
	
}

__global__ void convert_kernel(unsigned int *bins32, uint8_t *bins8,
    unsigned int num_bins) {

	int globalId = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	
	bins8[globalId] = (bins32[globalId] > 255) ? 255 : bins32[globalId];




}

/******************************************************************************
Setup and invoke your kernel(s) in this function. You may also allocate more
GPU memory if you need to
*******************************************************************************/
void histogram(unsigned int* input, uint8_t* bins, unsigned int num_elements,
        unsigned int num_bins) {

    

    // Create 32 bit bins
    unsigned int *bins32;
    hipMalloc((void**)&bins32, num_bins * sizeof(unsigned int));
    hipMemset(bins32, 0, num_bins * sizeof(unsigned int));

    // Launch histogram kernel using 32-bit bins
    dim3 dim_grid, dim_block;
    dim_block.x = 512; dim_block.y = dim_block.z = 1;
    dim_grid.x = 30; dim_grid.y = dim_grid.z = 1;
    histogram_kernel<<<dim_grid, dim_block, num_bins*sizeof(unsigned int)>>>
        (input, bins32, num_elements, num_bins);

    // Convert 32-bit bins into 8-bit bins
    dim_block.x = 512;
    dim_grid.x = (num_bins - 1)/dim_block.x + 1;
    convert_kernel<<<dim_grid, dim_block>>>(bins32, bins, num_bins);

    // Free allocated device memory
    hipFree(bins32);

}


